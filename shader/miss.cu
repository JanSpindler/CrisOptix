#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <hip/hip_runtime.h>
#include <graph/Interaction.h>
#include <graph/trace.h>
#include <graph/restir/PrefixSearchPayload.h>

extern "C" __global__ void __miss__prefix_entry()
{
}

extern "C" __global__ void __miss__main()
{
	Interaction* si = GetPayloadDataPointer<Interaction>();

	const glm::vec3 world_ray_origin = cuda2glm(optixGetWorldRayOrigin());
	const glm::vec3 world_ray_dir = cuda2glm(optixGetWorldRayDirection());

	si->valid = false;
	si->inRayDir = world_ray_dir;
}

extern "C" __global__ void __miss__occlusion()
{
	SetOcclusionPayload(false);
}
