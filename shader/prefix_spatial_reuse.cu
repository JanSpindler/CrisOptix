#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix_device.h>
#include <util/glm_cuda.h>
#include <graph/LaunchParams.h>
#include <graph/restir/path_gen.h>
#include <util/pixel_index.h>
#include <graph/restir/prefix_reuse.h>

__constant__ LaunchParams params;

static __forceinline__ __device__ void PrefixSpatialReuse(const glm::uvec2& pixelCoord, PCG32& rng)
{
	// Assume: pixelCoord are valid

	// Get current prefix
	const size_t currPixelIdx = GetPixelIdx(pixelCoord, params);
	Reservoir<PrefixPath>& currPrefixRes = params.restir.prefixReservoirs[currPixelIdx];
	const PrefixPath& currPrefix = currPrefixRes.sample;

	// Exit if current prefix is invalid or not fit for reuse
	if (!currPrefix.valid || currPrefix.len < params.restir.minPrefixLen) { return; }

	// Select random neighbor
	static constexpr uint32_t kernelRadius = 2;
	const uint32_t randX = rng.NextUint32() % (2 * kernelRadius + 1);
	const uint32_t randY = rng.NextUint32() % (2 * kernelRadius + 1);
	const glm::uvec2 neighPixelCoord = pixelCoord + glm::uvec2(randX, randY) - glm::uvec2(kernelRadius, kernelRadius);

	// Exit if neighbor is current pixel
	if (neighPixelCoord == pixelCoord) { return; }

	// Check if neighbor is on screen
	if (!IsPixelValid(neighPixelCoord, params)) { return; }

	// Get neighbor prefix reservoir
	const Reservoir<PrefixPath>& neighPrefixRes = params.restir.prefixReservoirs[GetPixelIdx(neighPixelCoord, params)];
	const PrefixPath& neighPrefix = neighPrefixRes.sample;

	// Prefix reuse
	const SurfaceInteraction& primaryInteraction = params.restir.restirGBuffers[currPixelIdx].primaryInteraction;
	PrefixReuse(currPrefixRes, neighPrefixRes, primaryInteraction, rng, params);
}

extern "C" __global__ void __raygen__prefix_spatial_reuse()
{
	// Sanity check
	if (!params.enableRestir || !params.restir.prefixEnableSpatial) { return; }

	//
	const glm::uvec3 launchIdx = cuda2glm(optixGetLaunchIndex());
	const glm::uvec3 launchDims = cuda2glm(optixGetLaunchDimensions());
	const glm::uvec2 pixelCoord = glm::uvec2(launchIdx);

	// Exit if invalid launch idx
	if (launchIdx.x >= params.width || launchIdx.y >= params.height || launchIdx.z >= 1)
	{
		return;
	}

	// Init RNG
	// TODO: Other seed for each raygen
	const uint32_t pixelIdx = launchIdx.y * launchDims.x + launchIdx.x;
	const uint64_t seed = SampleTEA64(pixelIdx, params.random);
	PCG32 rng(seed);

	//
	PrefixSpatialReuse(pixelCoord, rng);
}
