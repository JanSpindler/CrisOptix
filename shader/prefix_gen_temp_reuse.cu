#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix_device.h>
#include <util/glm_cuda.h>
#include <graph/LaunchParams.h>
#include <graph/restir/path_gen.h>

__constant__ LaunchParams params;

static __forceinline__ __device__ void PrefixGen(
	Reservoir<PrefixPath>& prefixRes,
	const glm::uvec2& pixelCoord,
	const glm::vec3& origin,
	const glm::vec3& dir,
	PCG32& rng)
{
	const PrefixPath prefix = TracePrefix(origin, dir, params.restir.minPrefixLen, 8, rng, params);
	if (!prefix.valid) { return; }

	const float pHat = GetLuminance(prefix.f);
	const float risWeight = pHat / prefix.p;
	const glm::vec3 fOverP = prefix.f / prefix.p;
	prefixRes.Update(prefix, risWeight, fOverP, rng);
}

static __forceinline__ __device__ void PrefixTempReuse(Reservoir<PrefixPath>& prefixRes)
{
}

extern "C" __global__ void __raygen__prefix_gen_temp_reuse()
{
	//
	const glm::uvec3 launchIdx = cuda2glm(optixGetLaunchIndex());
	const glm::uvec3 launchDims = cuda2glm(optixGetLaunchDimensions());
	const glm::uvec2 pixelCoord = glm::uvec2(launchIdx);

	// Exit if invalid launch idx
	if (launchIdx.x >= params.width || launchIdx.y >= params.height || launchIdx.z >= 1)
	{
		return;
	}

	// Init RNG
	const uint32_t pixelIdx = launchIdx.y * launchDims.x + launchIdx.x;
	const uint64_t seed = SampleTEA64(pixelIdx, params.random);
	PCG32 rng(seed);

	// Init radiance with 0
	glm::vec3 outputRadiance(0.0f);

	// Spawn camera ray
	glm::vec3 origin(0.0f);
	glm::vec3 dir(0.0f);
	glm::vec2 uv = (glm::vec2(launchIdx) + rng.Next2d()) / glm::vec2(params.width, params.height);
	uv = 2.0f * uv - 1.0f; // [0, 1] -> [-1, 1]
	SpawnCameraRay(params.cameraData, uv, origin, dir);

	if (params.enableRestir)
	{
		Reservoir<PrefixPath> prefixRes{};
		PrefixGen(prefixRes, pixelCoord, origin, dir, rng);
		PrefixTempReuse(prefixRes);
	}
	else
	{
		outputRadiance = TraceCompletePath(origin, dir, 8, 8, rng, params);
	}

	// Store radiance output
	if (params.enableAccum)
	{
		const glm::vec3 oldVal = params.outputBuffer[pixelIdx];
		const float blendFactor = 1.0f / static_cast<float>(params.frameIdx + 1);
		params.outputBuffer[pixelIdx] = blendFactor * outputRadiance + (1.0f - blendFactor) * oldVal;
	}
	else
	{
		params.outputBuffer[pixelIdx] = outputRadiance;
	}
}
