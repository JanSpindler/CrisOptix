#include "hip/hip_runtime.h"
#include <optix.h>
#include <hip/hip_runtime.h>
#include <util/glm_cuda.h>
#include <optix_device.h>
#include <graph/LaunchParams.h>
#include <util/random.h>
#include <util/shader_util.h>
#include <graph/Interaction.h>
#include <graph/trace.h>

static constexpr uint32_t MAX_TRACE_OPS = 128;
static constexpr uint32_t MAX_TRACE_DEPTH = 8;

__constant__ LaunchParams params;

struct Ray
{
	glm::vec3 origin;
	glm::vec3 dir;
	glm::vec3 throughput; // Contribution to final radiance value of pixel
	uint32_t depth;
};

extern "C" __global__ void __miss__main()
{
	SurfaceInteraction* si = GetPayloadDataPointer<SurfaceInteraction>();

	const glm::vec3 world_ray_origin = cuda2glm(optixGetWorldRayOrigin());
	const glm::vec3 world_ray_dir = cuda2glm(optixGetWorldRayDirection());
	const float tmax = optixGetRayTmax();

	si->valid = false;
	si->inRayDir = world_ray_dir;
	si->inRayDist = tmax;
}

extern "C" __global__ void __raygen_main()
{
	const glm::uvec3 launchIdx = cuda2glm(optixGetLaunchIndex());
	const glm::uvec3 launchDims = cuda2glm(optixGetLaunchDimensions());

	if (launchIdx.x >= params.width || launchIdx.y >= params.height || launchIdx.z >= 1)
	{
		return;
	}

	const uint32_t pixelIdx = launchIdx.y * launchDims.x + launchIdx.x;
	const uint64_t seed = SampleTEA64(pixelIdx, 1);
	PCG32 rng(seed);

	glm::vec3 outputRadiance(0.0f);

	bool nextRayValid = true;
	Ray nextRay{};
	{
		glm::vec2 uv = (glm::vec2(launchIdx) + rng.Next2d()) / glm::vec2(params.width, params.height);
		uv = 2.0f * uv - 1.0f; // [0, 1] -> [-1, 1]
		SpawnCameraRay(params.cameraData, uv, nextRay.origin, nextRay.dir);
		nextRay.throughput = glm::vec3(1);
		nextRay.depth = 0;
	}

	for (uint32_t traceIdx = 0; traceIdx < MAX_TRACE_OPS; ++traceIdx)
	{
		if (!nextRayValid) { break; }

		Ray currentRay = nextRay;
		nextRayValid = false;

		SurfaceInteraction interaction{};
		TraceWithDataPointer(
			params.traversableHandle, 
			currentRay.origin, 
			currentRay.dir, 
			1e-3, 
			1e16, 
			params.surfaceTraceParams, 
			&interaction);

		if (!interaction.valid) { continue; }

		// TODO: Emitter
		outputRadiance = glm::vec3(1.0f);

		if (currentRay.depth >= MAX_TRACE_DEPTH) { continue; }
	}

	params.outputBuffer[pixelIdx] = outputRadiance;
}
