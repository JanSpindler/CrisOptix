#include "hip/hip_runtime.h"
#include <optix.h>
#include <hip/hip_runtime.h>
#include <util/glm_cuda.h>
#include <optix_device.h>

__global__ void __raygen_main()
{
	const glm::uvec3 launchIdx = cuda2glm(optixGetLaunchIndex());
	const glm::uvec3 launchDims = cuda2glm(optixGetLaunchDimensions());
}
