#include "hip/hip_runtime.h"
#include <optix.h>
#include <hip/hip_runtime.h>
#include <util/glm_cuda.h>
#include <optix_device.h>
#include <graph/LaunchParams.h>
#include <util/random.h>
#include <util/shader_util.h>
#include <graph/Interaction.h>
#include <graph/trace.h>
#include <graph/brdf.h>

static constexpr uint32_t MAX_TRACE_OPS = 32;
static constexpr uint32_t MAX_TRACE_DEPTH = 8;

__constant__ LaunchParams params;

struct Ray
{
	glm::vec3 origin;
	glm::vec3 dir;
	glm::vec3 throughput; // Contribution to final radiance value of pixel
	uint32_t depth;
};

extern "C" __global__ void __closesthit__mesh()
{
	// Get interaction by ptr
	SurfaceInteraction* si = GetPayloadDataPointer<SurfaceInteraction>();
	const MeshSbtData* sbtData = reinterpret_cast<const MeshSbtData*>(optixGetSbtDataPointer());
	si->meshSbtData = sbtData;

	// Fill ray info
	const glm::vec3 worldRayOrigin = cuda2glm(optixGetWorldRayOrigin());
	const glm::vec3 worldRayDir = cuda2glm(optixGetWorldRayDirection());
	const float tMax = optixGetRayTmax();

	// Fill basic interaction info
	si->inRayDir = worldRayDir;
	si->inRayDist = tMax;
	si->valid = true;

	// Get primitive data
	const uint32_t primIdx = optixGetPrimitiveIndex();
	const glm::vec2 baryCoord = cuda2glm(optixGetTriangleBarycentrics());

	// Indices of triangle vertices in the mesh
	glm::uvec3 tri = glm::uvec3(0u);

	// Indices stored as 32-bit unsigned integers
	//const glm::u32vec3* indices = reinterpret_cast<glm::u32vec3*>(mesh_data->indices.data);
	//tri = glm::uvec3(indices[primIdx]);
}

extern "C" __global__ void __miss__main()
{
	SurfaceInteraction* si = GetPayloadDataPointer<SurfaceInteraction>();

	const glm::vec3 world_ray_origin = cuda2glm(optixGetWorldRayOrigin());
	const glm::vec3 world_ray_dir = cuda2glm(optixGetWorldRayDirection());
	const float tmax = optixGetRayTmax();

	si->valid = false;
	si->inRayDir = world_ray_dir;
	si->inRayDist = tmax;
}

extern "C" __global__ void __miss__occlusion()
{
	SetOcclusionPayload(false);
}

extern "C" __global__ void __raygen__main()
{
	const glm::uvec3 launchIdx = cuda2glm(optixGetLaunchIndex());
	const glm::uvec3 launchDims = cuda2glm(optixGetLaunchDimensions());

	if (launchIdx.x >= params.width || launchIdx.y >= params.height || launchIdx.z >= 1)
	{
		return;
	}

	const uint32_t pixelIdx = launchIdx.y * launchDims.x + launchIdx.x;
	const uint64_t seed = SampleTEA64(pixelIdx, 1);
	PCG32 rng(seed);

	glm::vec3 outputRadiance(0.0f);

	bool nextRayValid = true;
	Ray nextRay{};
	{
		glm::vec2 uv = (glm::vec2(launchIdx) + rng.Next2d()) / glm::vec2(params.width, params.height);
		uv = 2.0f * uv - 1.0f; // [0, 1] -> [-1, 1]
		SpawnCameraRay(params.cameraData, uv, nextRay.origin, nextRay.dir);
		nextRay.throughput = glm::vec3(1);
		nextRay.depth = 0;
	}

	//params.outputBuffer[pixelIdx] = glm::vec3(0.5f);
	//return;

	for (uint32_t traceIdx = 0; traceIdx < MAX_TRACE_OPS; ++traceIdx)
	{
		if (!nextRayValid) { break; }

		Ray currentRay = nextRay;
		nextRayValid = false;

		SurfaceInteraction interaction{};
		TraceWithDataPointer<SurfaceInteraction>(
			params.traversableHandle, 
			currentRay.origin, 
			currentRay.dir, 
			1e-3, 
			1e16, 
			params.surfaceTraceParams, 
			&interaction);

		if (!interaction.valid) { continue; }

		const glm::vec3 dirLightDir(0.0f, 0.0f, 1.0f);
		const BrdfResult brdfResult = optixDirectCall<BrdfResult, const SurfaceInteraction&, const glm::vec3&>(
			interaction.meshSbtData->evalMaterialSbtIdx, 
			interaction, 
			dirLightDir);
		outputRadiance = brdfResult.brdfResult;

		if (currentRay.depth >= MAX_TRACE_DEPTH) { continue; }
	}

	params.outputBuffer[pixelIdx] = outputRadiance;
}
