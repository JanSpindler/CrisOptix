#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix_device.h>
#include <graph/LaunchParams.h>
#include <util/glm_cuda.h>
#include <util/pixel_index.h>

__constant__ LaunchParams params;

extern "C" __global__ void __intersection__prefix_entry()
{
	printf("Prefix Entry Intersection\n");
}

extern "C" __global__ void __raygen__final_gather()
{
	//
	const glm::uvec3 launchIdx = cuda2glm(optixGetLaunchIndex());
	const glm::uvec3 launchDims = cuda2glm(optixGetLaunchDimensions());
	const glm::uvec2 pixelCoord = glm::uvec2(launchIdx);
	const size_t pixelIdx = GetPixelIdx(pixelCoord, params);

	// Exit if invalid launch idx
	if (launchIdx.x >= params.width || launchIdx.y >= params.height || launchIdx.z >= 1)
	{
		return;
	}

	// Init RNG
	PCG32& rng = params.restir.restirGBuffers[pixelIdx].rng;

	// Get prefix and suffix from this pixels restir
	const PrefixPath& prefix = params.restir.prefixReservoirs[pixelIdx].sample;
	const SuffixPath& suffix = params.restir.suffixReservoirs[pixelIdx].sample;

	// Display complete path contribution
	glm::vec3 outputRadiance(0.0f);
	if (prefix.valid)
	{
		if (prefix.nee)
		{
			outputRadiance = prefix.f / prefix.p;
		}
		else if (suffix.valid)
		{
			outputRadiance = (prefix.f * suffix.f) / (prefix.p * suffix.p);
		}
	}

	// Accum
	if (params.enableAccum)
	{
		const glm::vec3 oldVal = params.outputBuffer[pixelIdx];
		const float blendFactor = 1.0f / static_cast<float>(params.frameIdx + 1);
		params.outputBuffer[pixelIdx] = blendFactor * outputRadiance + (1.0f - blendFactor) * oldVal;
	}
	else
	{
		params.outputBuffer[pixelIdx] = outputRadiance;
	}
}
