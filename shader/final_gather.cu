#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix_device.h>
#include <graph/LaunchParams.h>
#include <util/glm_cuda.h>
#include <util/pixel_index.h>
#include <graph/trace.h>
#include <graph/restir/PrefixSearchPayload.h>
#include <graph/restir/path_gen.h>
#include <cuda/std/tuple>

__constant__ LaunchParams params;

extern "C" __global__ void __intersection__prefix_entry()
{
	// Get pixel index of hit
	const uint32_t neighPixelIdx = optixGetPrimitiveIndex();

	// Get payload
	PrefixSearchPayload* payload = GetPayloadDataPointer<PrefixSearchPayload>();

	// Check if radius is truly as desired
	const glm::vec3 queryPos = cuda2glm(optixGetWorldRayOrigin());
	const glm::vec3& neighPos = params.restir.prefixReservoirs[neighPixelIdx].sample.lastInteraction.pos;
	const float distance = glm::distance(queryPos, neighPos);
	if (distance > params.restir.gatherRadius) { return; }

	// Build neighbor
	const PrefixNeighbor neigh(neighPixelIdx, distance);

	// Store neigh pixel idx
	const uint32_t k = params.restir.gatherM - 1;
	const uint32_t offset = payload->pixelIdx * k;

	// If neigh pixel idx buffer not full
	if (payload->neighCount < k)
	{
		// Append neigh pixel idx to buffer
		params.restir.prefixNeighbors[offset + payload->neighCount] = neigh;

		// Inc neigh count
		++payload->neighCount;

		// Find stored neigh with largest distance
		if (payload->neighCount == k)
		{
			payload->FindLargestDist(params);
		}
	}
	// If neigh pixel idx buffer is full AND the distance of new neigh is lower than the max distance so far
	else if (distance < payload->maxNeighDist)
	{
		params.restir.prefixNeighbors[offset + payload->maxDistNeighIdx] = neigh;
		payload->FindLargestDist(params);
	}
}

static __forceinline__ __device__ glm::vec3 GetPathContribution(
	const PrefixPath& prefix, 
	const SuffixPath& suffix,
	const float prefixUcw,
	const float suffixUcw)
{
	glm::vec3 output(0.0f);

	if (prefix.valid)
	{
		if (prefix.nee)
		{
			output = prefix.f * prefixUcw;;
		}
		else if (suffix.valid)
		{
			output = prefix.f * suffix.f * prefixUcw * suffixUcw;
		}
	}

	if (glm::any(glm::isinf(output) || glm::isnan(output))) { return glm::vec3(0.0f); }
	return output;
}

static __forceinline__ __device__ cuda::std::pair<glm::vec3, float> ShiftSuffix(const PrefixPath& prefix, const SuffixPath& suffix)
{
	// Trace occlusion
	const bool occluded = TraceOcclusion(
		params.traversableHandle,
		prefix.lastInteraction.pos,
		glm::normalize(suffix.reconInteraction.pos - prefix.lastInteraction.pos),
		1e-3f,
		glm::distance(suffix.reconInteraction.pos, prefix.lastInteraction.pos),
		params.occlusionTraceParams);
	if (occluded) { return { glm::vec3(0.0f), 0.0f }; }

	return { suffix.f, 1.0f / suffix.p };
}

static __forceinline__ __device__ glm::vec3 GetRadiance(const glm::uvec3& launchIdx, const size_t pixelIdx, PCG32& rng)
{
	// Get prefix and suffix from this pixels restir
	const PrefixPath& prefix = params.restir.prefixReservoirs[pixelIdx].sample;
	const SuffixPath& suffix = params.restir.suffixReservoirs[pixelIdx].sample;

	// Exit if prefix is invalid
	if (!params.restir.restirGBuffers[pixelIdx].primaryInteraction.valid)
	{
		return glm::vec3(0.0f);
	}

	// Init empty output radiance
	glm::vec3 outputRadiance(0.0f);

	// Final gather
	// Spawn camera ray
	glm::vec3 origin(0.0f);
	glm::vec3 dir(0.0f);
	glm::vec2 uv = (glm::vec2(launchIdx) + rng.Next2d()) / glm::vec2(params.width, params.height);
	uv = 2.0f * uv - 1.0f; // [0, 1] -> [-1, 1]
	SpawnCameraRay(params.cameraData, uv, origin, dir);

	// K = M - 1
	float canonSuffixMisWeight = 1.0f;
	const size_t k = params.restir.gatherM - 1;
	if (k > 0)
	{
		for (size_t prefixIdx = 0; prefixIdx < params.restir.gatherN; ++prefixIdx)
		{
			// Trace new prefix for pixel q
			SurfaceInteraction interaction{};
			const PrefixPath neighPrefix = prefixIdx == 0 ? prefix : TracePrefix(origin, dir, params.restir.minPrefixLen, 8, interaction, rng, params);
			if (!neighPrefix.valid) { continue; }

			// Find k neighboring prefixes in world space
			static constexpr float EPSILON = 1e-16;
			PrefixSearchPayload prefixSearchPayload(pixelIdx);
			TraceWithDataPointer<PrefixSearchPayload>(
				params.restir.prefixEntriesTraversHandle,
				neighPrefix.lastInteraction.pos,
				glm::vec3(EPSILON),
				0.0f,
				EPSILON,
				params.restir.prefixEntriesTraceParams,
				&prefixSearchPayload);
			const uint32_t neighCount = prefixSearchPayload.neighCount;
			const float misWeight = 1.0f / static_cast<float>(neighCount + 1.0f);

			// Set mis weight for canonical suffix
			if (prefixIdx == 0)
			{
				canonSuffixMisWeight = misWeight;
			}

			// Track prefix stats
			if (params.restir.trackPrefixStats)
			{
				atomicMin(&params.restir.prefixStats[0].minNeighCount, neighCount);
				atomicMax(&params.restir.prefixStats[0].maxNeighCount, neighCount);
				atomicAdd(&params.restir.prefixStats[0].totalNeighCount, neighCount);
			}

			// Borrow their suffixes and gather path contributions
			for (size_t suffixIdx = 0; suffixIdx < neighCount; ++suffixIdx)
			{
				// Assume: Neighbor prefix and suffix are valid

				// Get suffix
				const uint32_t suffixPixelIdx = params.restir.prefixNeighbors[k * pixelIdx + suffixIdx].pixelIdx;
				const SuffixPath& neighSuffix = params.restir.suffixReservoirs[suffixPixelIdx].sample;

				// Shift suffix
				const cuda::std::pair<glm::vec3, float> shiftedSuffix = ShiftSuffix(neighPrefix, neighSuffix);
				const glm::vec3& shiftedF = shiftedSuffix.first;
				const float ucwSuffix = shiftedSuffix.second;

				// Calc path contribution
				const glm::vec3 pathContrib = glm::max(glm::vec3(0.0f), neighPrefix.f * shiftedF);

				// Calc ucw
				const float ucw = ucwSuffix / neighPrefix.p;

				// Gather
				outputRadiance += misWeight * pathContrib * ucw;
			}
		}

		outputRadiance /= static_cast<float>(params.restir.gatherN);
	}

	// Add canon suffix contrib
	outputRadiance += canonSuffixMisWeight * TraceCompletePath(origin, dir, 8, 8, rng, params);

	if (glm::any(glm::isnan(outputRadiance) || glm::isinf(outputRadiance))) { outputRadiance = glm::vec3(0.0f); }

	return outputRadiance;
}

extern "C" __global__ void __raygen__final_gather()
{
	//
	const glm::uvec3 launchIdx = cuda2glm(optixGetLaunchIndex());
	const glm::uvec3 launchDims = cuda2glm(optixGetLaunchDimensions());
	const glm::uvec2 pixelCoord = glm::uvec2(launchIdx);
	const size_t pixelIdx = GetPixelIdx(pixelCoord, params);

	// Exit if invalid launch idx
	if (launchIdx.x >= params.width || launchIdx.y >= params.height || launchIdx.z >= 1)
	{
		return;
	}

	// Init RNG
	PCG32& rng = params.restir.restirGBuffers[pixelIdx].rng;

	// Accum
	const glm::vec3 outputRadiance = GetRadiance(launchIdx, pixelIdx, rng);
	if (params.enableAccum)
	{
		const glm::vec3 oldVal = params.outputBuffer[pixelIdx];
		const float blendFactor = 1.0f / static_cast<float>(params.frameIdx + 1);
		params.outputBuffer[pixelIdx] = blendFactor * outputRadiance + (1.0f - blendFactor) * oldVal;
	}
	else
	{
		params.outputBuffer[pixelIdx] = outputRadiance;
	}
}
