#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix_device.h>
#include <graph/LaunchParams.h>
#include <util/glm_cuda.h>
#include <util/pixel_index.h>

__constant__ LaunchParams params;

extern "C" __global__ void __raygen__final_gather()
{
	//
	const glm::uvec3 launchIdx = cuda2glm(optixGetLaunchIndex());
	const glm::uvec3 launchDims = cuda2glm(optixGetLaunchDimensions());
	const glm::uvec2 pixelCoord = glm::uvec2(launchIdx);
	const size_t pixelIdx = GetPixelIdx(pixelCoord, params);

	// Exit if invalid launch idx
	if (launchIdx.x >= params.width || launchIdx.y >= params.height || launchIdx.z >= 1)
	{
		return;
	}

	// Init RNG
	const uint64_t seed = SampleTEA64(pixelIdx, params.random);
	PCG32 rng(seed);

	//
	const PrefixPath& prefix = params.restir.prefixReservoirs[pixelIdx].sample;
	const SuffixPath& suffix = params.restir.suffixReservoirs[pixelIdx].sample;

	glm::vec3 outputRadiance(0.0f);
	if (prefix.valid)
	{
		if (suffix.valid)
		{
			outputRadiance = (prefix.f * suffix.f) / (prefix.p * suffix.p);
		}
		else if (prefix.nee)
		{
			outputRadiance = prefix.f / prefix.p;
		}
	}

	params.outputBuffer[pixelIdx] = outputRadiance;
}
