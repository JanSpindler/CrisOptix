#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix_device.h>
#include <graph/LaunchParams.h>
#include <util/glm_cuda.h>
#include <util/pixel_index.h>

__constant__ LaunchParams params;

extern "C" __global__ void __raygen__prefix_store_entries()
{
	//
	const glm::uvec3 launchIdx = cuda2glm(optixGetLaunchIndex());
	const glm::uvec3 launchDims = cuda2glm(optixGetLaunchDimensions());
	const glm::uvec2 pixelCoord = glm::uvec2(launchIdx);

	// Exit if invalid launch idx
	if (launchIdx.x >= params.width || launchIdx.y >= params.height || launchIdx.z >= 1)
	{
		return;
	}

	// Get prefix and suffix
	const size_t pixelIdx = GetPixelIdx(pixelCoord, params);
	const PrefixPath& prefix = params.restir.prefixReservoirs[pixelIdx * 2 + params.restir.frontBufferIdx].sample;
	const SuffixPath& suffix = params.restir.suffixReservoirs[pixelIdx * 2 + params.restir.frontBufferIdx].sample;

	// Get last prefix interaction
	const Interaction lastPrefixInt(prefix.lastInt, params.transforms);

	// Store aabb
	OptixAabb& aabb = params.restir.prefixEntryAabbs[pixelIdx];
	if (prefix.IsValid() && suffix.IsValid() && lastPrefixInt.valid)
	{
		const float radius = params.restir.gatherRadius;
		const glm::vec3& pos = lastPrefixInt.pos;

		aabb.minX = pos.x - radius;
		aabb.minY = pos.y - radius;
		aabb.minZ = pos.z - radius;
		aabb.maxX = pos.x + radius;
		aabb.maxY = pos.y + radius;
		aabb.maxZ = pos.z + radius;

		//printf("%f, %f, %f, %f, %f, %f\n", aabb.minX, aabb.maxX, aabb.minY, aabb.maxY, aabb.minZ, aabb.maxZ);
	}
	else
	{
		aabb = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };
	}
}
