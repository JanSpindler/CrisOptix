#include "hip/hip_runtime.h"
#include <kernel/tonemap.h>
#include <hip/hip_runtime.h>
#define GLM_ENABLE_EXPERIMENTAL
#include <glm/gtx/compatibility.hpp>
#include <util/math.h>

template <typename T>
static __forceinline__ __host__ __device__ T ApplySrgbGamma(const T& linearColor)
{
	// Proper sRGB curve...
	auto cond = glm::lessThan(linearColor, T(0.0031308f));
	auto if_true = 12.92f * linearColor;
	auto if_false = 1.055f * glm::pow(linearColor, T(1.0f / 2.4f)) - 0.055f;
	return glm::lerp(if_false, if_true, T(cond));
	// return c <= 0.0031308f ? 12.92f * c : 1.055f * powf(c, 1.0f/2.4f) - 0.055f;
}

static __forceinline__ __host__ __device__ glm::u8vec3 LinearToSrgb(const glm::vec3& linearColor)
{
	return static_cast<glm::u8vec3>(glm::clamp(ApplySrgbGamma(linearColor), 0.0f, 1.0f) * 255.0f);
}

static __global__ void ToneMappingKernel(const CuBufferView<glm::vec3> inputHdr, CuBufferView<glm::u8vec3> outputLdr)
{
	const uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= inputHdr.count)
	{
		return;
	}

	outputLdr[idx] = LinearToSrgb(inputHdr[idx]);
	//outputLdr[idx] = glm::u8vec3(100);
}

void ToneMapping(const CuBufferView<glm::vec3>& inputHdr, CuBufferView<glm::u8vec3>& outputLdr)
{
	const uint32_t blockSize = 512;
	const uint32_t blockCount = CeilDiv(inputHdr.count, blockSize);
	ToneMappingKernel<<<blockSize, blockCount>>>(inputHdr, outputLdr);
}
