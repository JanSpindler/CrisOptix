#include <iostream>
#include <Window.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <optix_host.h>
#include <cuda_gl_interop.h>
#include <optix.h>
#include <optix_stubs.h>
#include <optix_function_table_definition.h>
#include <custom_assert.h>

OptixDeviceContext optixContext = nullptr;

void InitOptix()
{
    if (hipFree(0) != hipSuccess) { exit(1); }
    if (optixInit() != OPTIX_SUCCESS) { exit(1); }

    const OptixDeviceContextOptions optixContextOptions{};
    optixContext = nullptr;
    ASSERT_OPTIX(optixDeviceContextCreate(0, &optixContextOptions, &optixContext));
}

int main()
{
    std::cout << "Hello there" << std::endl;

    Window::Init(800, 600, false, "CrisOptix");
    InitOptix();

    while (!Window::IsClosed())
    {
        Window::Update();
    }

    Window::Destroy();

    return 0;
}
