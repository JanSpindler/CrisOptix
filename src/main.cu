#include <iostream>
#include <Window.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <optix_host.h>
#include <cuda_gl_interop.h>
#include <optix.h>
#include <optix_stubs.h>
#include <optix_function_table_definition.h>
#include <custom_assert.h>

OptixDeviceContext optixContext = nullptr;

void InitOptix()
{
    if (hipFree(0) != hipSuccess) { exit(1); }
    if (optixInit() != OPTIX_SUCCESS) { exit(1); }

    const OptixDeviceContextOptions optixContextOptions{};
    optixContext = nullptr;
    ASSERT_OPTIX(optixDeviceContextCreate(0, &optixContextOptions, &optixContext));
}

int main()
{
    std::cout << "Hello there" << std::endl;

    Window::Init(800, 600, false, "CrisOptix");
    InitOptix();

    OutputBuffer<glm::u8vec3> outputBuffer(800, 600);

    DeviceBuffer<glm::vec3> hdrBuffer(800 * 600);

    while (!Window::IsClosed())
    {
        //
        outputBuffer.MapCuda();

        // Trace rays
        // TODO: trace rays

        // Tone mapping

        // Render to window
        Window::Update();
    }

    Window::Destroy();

    return 0;
}
